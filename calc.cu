#include "hip/hip_runtime.h"
#include "calc.h"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void movingAvg(int n, int numDays, double* in, double* out){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		out[i] = 0;
		
		if(i>=numDays-1){
			for(int j = i-numDays+1; j <= i; j++){
				out[i] += in[j];
				// if(i==numDays-1)
					// printf("%d %f %f\n", i, in[j], out[i]);
			}
			out[i] /= numDays;
		}
	}
}

__global__ void deltas(int n, double* in, double* out){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		if(i==0) out[i] = 0;
		else{
			out[i] = in[i] - in[i-1];
		}
	}
}

__global__ void stdDev(int n, int period, double* x, double* std){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i <= period){
		std[i] = 0;
	}
	if(i < n && i > period){
		double average = 0;
		for(int j = i-period+1; j <= i; j++){
			average += x[j];
		}
		average /= period;
		double deviation = 0;
		for(int j = i-period+1; j <= i; j++){
			deviation += (x[j] - average) * (x[j] - average);
		}
		deviation /= period;
		std[i] = __dsqrt_rn(deviation);
		// 
	}
	// if(i==n)
	// 	for(int j = 0; j < n; j++)
	// 		printf("%d %f\n", j, std[j]);
}

__global__ void normalize(int* n, double* ave, double* in){
	// auto g = this_grid();
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	// if(i == 0)
	// 	for(int j = 0; j < n; j++)
	// 		printf("%d %f\n", j, in[j]);
	// __syncthreads();
	// if(i == n)
	// 	for(int j = 0; j < n; j++)
	// 		printf("%d %f\n", j, in[j]);
	// // // if(i<n)
	// // // 	printf("%d %f\n", i, in[i]);
	// __syncthreads();
	if(i == *n){
		// printf("%p\n", in);
		*ave = 0;
		for(int j = 0; j < *n; j++){
			// printf("%d %p %f %f\n", j, ave, *ave, *(in+j));
			// printf("%d %f\n", j, in[j]);
			(*ave) = (*ave) + in[j];
		}
		(*ave) = (*ave) / *n;
		// printf("%i\n", g.size());
	}

	__syncthreads();

	// g.sync();

	if(i < *n){
		// printf("%d %f\n", i, *ave);
		in[i] /= (*ave);
	}
}

__global__ void pearson(int nx, double* x, int* xDates, int ny, double* y, int* yDates, double* r, double* num, double* Xden, double* Yden, double* aveX, double* aveY){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	auto g = this_grid();
	// __shared__ double aveX;
	// __shared__ double aveY;

	if(i==nx){
		double sumX = 0, sumY = 0;
		for(int j = 0; j < nx; j++){
			sumX += x[j];
		}
		for(int j = 0; j < ny; j++){
			sumY += y[j];
		}
		*aveX = sumX/nx;
		*aveY = sumY/ny;
		// __shared__ double num[n];
		// __shared__ double Xden[n];
		// __shared__ double Yden[n];
		// printf("%f %f\n", aveX, aveY);
	}

	// __syncthreads();
	g.sync();
	if(i<nx){
		// printf("%s %d %d %d\n", "cuda", i, nx, ny);
		int j;
		for(j = 0; j < ny; j++){
			if(xDates[i] > yDates[j] && xDates[i] < yDates[j+1]){
				break;
			}
		}
		// printf("%d %d %d %d\n", i, xDates[i], yDates[j], j);
		// printf("%f %f\n", x[i], y[j]);
		num[i] = (x[i]-*aveX)*(y[j]-*aveY);
		Xden[i] = ((x[i]-*aveX)*(x[i]-*aveX));
		Yden[i] = ((y[j]-*aveY)*(y[j]-*aveY));
	}

	// __syncthreads();
	g.sync();

	if(i==nx){
		double numerator = 0, d1 = 0, d2 = 0;
		for(int j = 0; j < nx; j++){
			numerator += num[j];
			d1 += Xden[j];
			d2 += Yden[j];
		}
		*r = numerator/((__dsqrt_rn(d1))*(__dsqrt_rn(d2)));
	}
}

// void calcAverage(int numPoints, int size, double* input, double* output, double* delta);

double* calcPearson(int nx, double* x, int* xDates, int ny, double* y, int* yDates){
	double* correlation = new double;
	double* d_r, *d_aveX, *d_aveY;

	hipMalloc((void**)&d_r, sizeof(double));
	hipMalloc((void**)&d_aveX, sizeof(double));
	hipMalloc((void**)&d_aveY, sizeof(double));

	double* d_numerator, *d_Xden, *d_Yden;

	

	if(nx > ny){
		gpuErrchk(hipMalloc((void**)&d_numerator, nx*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Xden, nx*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Yden, nx*sizeof(double)));
		pearson<<<(nx)/1024 + 1, 1024>>>(nx, x, xDates, ny, y, yDates, d_r, d_numerator, d_Xden, d_Yden, d_aveX, d_aveY);
	}
	else{
		gpuErrchk(hipMalloc((void**)&d_numerator, ny*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Xden, ny*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Yden, ny*sizeof(double)));
		pearson<<<(ny)/1024 + 1, 1024>>>(ny, y, yDates, nx, x, xDates, d_r, d_numerator, d_Xden, d_Yden, d_aveX, d_aveY);
	}

	hipMemcpy(correlation, d_r, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_numerator);
	hipFree(d_Xden);
	hipFree(d_Yden);
	hipFree(d_r);
	hipFree(d_aveX);
	hipFree(d_aveY);

	return correlation;
}

int parseDate(char* date){
	char* year, *month, *day;
	bool first = false;
	year = date;
	for(int i = 0; date[i] != '\0'; i++){
		if(date[i] == '-'){
			date[i] = '\0';
			if(!first){
				first = true;
				month = &date[i+1];
			}
			else{
				day = &date[i+1];
			}
		}
	}
	// cout<<year<<endl;
	// cout<<month<<endl;
	// cout<<day<<endl;
	return stoi(year)*365 + stoi(month)*30 + stoi(day);
}

__global__ void optionPrice(double* stockPrices, int numDays, double* strikes, double* maturity, bool* call, double* optionPrices, int numOptions){
	// printf("Called OptionPrice\n");
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	double R = 1.0202;
	__shared__ double deviation;
	if(threadIdx.x == 0){
		double average = 0;
		for(int j = 0; j <= numDays; j++){
			average += stockPrices[j];
		}
		average /= numDays;
		// printf("numdays: %d average: %f\n", numDays, average);
		deviation = 0;
		for(int j = 0; j <= numDays; j++){
			double in = (stockPrices[j] - average) * (stockPrices[j] - average);
			deviation += in;
			// printf("%f\n", in);
		}
		deviation /= numDays-1;
		// deviation = 
		// volatility = __dsqrt_rn(deviation);
		deviation = __dsqrt_rn(deviation);
		deviation *= __dsqrt_rn(numDays);
		deviation /= average;
		// printf("%f\n", deviation);
	}

	__syncthreads();
	// printf("%d\n", numOptions);
	if (i < numOptions){
		if(call[i]){
			double priceUp = stockPrices[numDays-1] + stockPrices[numDays-1] * deviation * maturity[i];
			double priceDown = stockPrices[numDays-1] - stockPrices[numDays-1] * deviation * maturity[i];
			if(priceDown < 0) priceDown = 0;
			// printf("%d %f %f %c\n", i, strikes[i], maturity[i], call[i]);
			// printf("+%f -%f\n", priceUp, priceDown);
			double maxUp = priceUp - strikes[i];
			if(maxUp < 0) maxUp = 0;
			double maxDown = priceDown - strikes[i];
			if(maxDown < 0) maxDown = 0;
			double valPrice = (maxUp-maxDown)/(priceUp-priceDown);
			optionPrices[i] = (stockPrices[numDays-1] * valPrice) + ((maxUp - (priceUp * valPrice))*exp(R*maturity[i]));
			// printf("%d %f %f %f %f\n", i, stockPrices[numDays-1], maturity[i], strikes[i], optionPrices[i]);
		}
	}

	// __syncthreads();
	
}
// double* stockPrices, int numDays, double* strikes, double* maturity, bool* call, double* optionPrices, int numOptions
__global__ void launch(double** prices, int* sizes, int n, int* status, double** optionPrices, double** strikes, double** exp, bool** calls, int* numOptions){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	// printf("%d %d\n", i, n);
	if(i < n){
		printf("in kernel %d\n", sizes[i]);
		if(sizes[i] > 10){
			// printf("%d < %d\n", i, n);
			// printf("%d %d\n", i, prices[i][0]);
			unsigned int blocks = 512/sizes[i]+1, threads = 512;
			// const dim3 coopBlocks = {blocks, 1, 1};
			// const dim3 coopThreads = {threads, 1, 1};
			double* ten = new double[sizes[i]], *five = new double[sizes[i]];
			double* d1, *d5, *d10, *d2_5, *d2_10, *std;
			
			d1 = new double[sizes[i]];
			d5 = new double[sizes[i]];
			d10 = new double[sizes[i]];
			d2_5 = new double[sizes[i]];
			d2_10 = new double[sizes[i]];
			// do{
			std = new double[sizes[i]];
				// if(std == 0)
					// delete[] std;
			// }while(std == 0);
			// printf("%p %p %p %p %p %p\n", d1, d5, d10, d2_5, d2_10, std);
			// normstdev = new double[sizes[i]];
			double* ave = new double;
			// for(int j = 0; j < sizes[i]; j++){
			// 	printf("%d %f", i, prices[i][j]);
			// __syncthreads();
			hipDeviceSynchronize();
			movingAvg<<<blocks, threads>>>(sizes[i], 10, prices[i], five);
			hipDeviceSynchronize();
			movingAvg<<<blocks, threads>>>(sizes[i], 20, prices[i], ten);
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], prices[i], d1);
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], five, d5);
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], ten, d10);
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], d5, d2_5);
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], d10, d2_10);
			hipDeviceSynchronize();
			stdDev<<<blocks, threads>>>(sizes[i], 20, prices[i], std);
			hipDeviceSynchronize();
			if(sizes[i] >= 253){
				// printf("Attempting to Launch optionPrice\n");
				optionPrice<<<numOptions[i]/512+1, 512>>>(&(prices[i][sizes[i]-253]), 252, strikes[i], exp[i], calls[i], optionPrices[i], numOptions[i]);
			}			
			else{
				// printf("Attempting to Launch optionPrice with fewer than 253 options\n");
				optionPrice<<<numOptions[i]/512+1, 512>>>(prices[i], sizes[i], strikes[i], exp[i], calls[i], optionPrices[i], numOptions[i]);
			}
			hipDeviceSynchronize();

			// void* paramlist[3] = {(void*)&sizes[i], (void*)&ave, (void*)&stdev};
			// hipLaunchCooperativeKernel((void*)normalize, coopBlocks, coopThreads, paramlist);
			normalize<<<blocks, threads>>>(&sizes[i], ave, std);
			status[i] = 0;
			// __syncthreads();
			int index = sizes[i]-1;
			// if(i==821)
			// for(int index = 0; index < sizes[i]; index++)
			// 	printf("%d %d %f %f %f %f %f %f %f\n", i, index, prices[i][index], d1[index], five[index], d5[index], d2_5[index], ten[index], d10[index], d2_10[index], std[index]);
			if((d10[index] < 0.02 || d10[index] > -0.02) && d2_5[index] > 0 && d5[index] > 0 && std[index] < 2){
				status[i] = 1;
			}
			else if( ((d2_5[index] < 0 && (zero(d5[index]) || d5[index] < -0.002)) ) || (d2_10[index] < 0 && (zero(d10[index] || d10[index] < -.002))) ) {
				status[i] = 2;
			}
			else if(prices[i][index-1] < five[index-1] && prices[i][index] > five[index] && std[index] > 1){
				status[i] = 2;
			}
			// __syncthreads();
			hipDeviceSynchronize();
			delete[] ten;
			delete[] five;
			delete[] d1;
			delete[] d5;
			delete[] d10;
			delete[] d2_5;
			delete[] d2_10;
			delete[] std;
			delete ave;
		}
	}
}

int main(int argc, char** argv) {
	gpuErrchk(hipSetDevice(1));
	hipDeviceSynchronize();

	// CSVReader file = CSVReader(argv[1]);
	// CSVReader option = CSVReader(argv[1]);
	// cout<<argv[1]<<endl;
	// int on = 0;
	// // gpuErrchk(hipDevicePrimaryCtxGetState(1, nullptr, &on));
	// if(on){
	// 	cout<<"Context initialized"<<endl;
	// }
	vector<char*>* symbols = new vector<char*>();

	if(argc <= 1)
		listDir("./proc", symbols);
	else{
		for (int i = 1; i < argc; i++){
			char* temp = new char[strlen(argv[i])+5];
			strcpy(temp, argv[i]);
			cout<<"Reading "<<temp<<endl;
			addStrToVec(strcat(temp,".csv"), symbols);
			delete [] temp;
		}

	}

	vector<AVData*> dataList = vector<AVData*>();
	vector<OptionData*> optionList = vector<OptionData*>();

	for(int i = 0; i < symbols->size(); i++){
		char* dirName = new char[100];
		strcpy(dirName, "./proc/");
		// cout<<(*symbols)[i]<<endl;
		AVData* temp = new AVData(strcat(dirName, (*symbols)[i]));
		dataList.push_back(temp);

		strcpy(dirName, "./options/");
		OptionData* temp2 = new OptionData(strcat(dirName, (*symbols)[i]));
		optionList.push_back(temp2);
	}

	double** d_prices, **prices = new double*[dataList.size()];
	int* d_pSizes, *pSizes = new int[dataList.size()];

	
	// cout<<wrapperSize<<' '<<cudaMem<<' '<<cudaMemTotal<<endl;
	// hipDeviceSynchronize();

	gpuErrchk(hipMalloc((void**)&d_pSizes, dataList.size()*sizeof(int)));
	// hipDeviceSynchronize();
	gpuErrchk(hipMalloc((void**)&d_prices, dataList.size()*sizeof(double*)));
	hipDeviceSynchronize();
	size_t totalSize = 0;
	// totalSize += dataList.size()*sizeof(int) + dataList.size()*sizeof(double*);
	for(int i = 0; i < dataList.size(); i++){
		
		dataList[i]->tokenize();
		// printf("%d %d %s\n", i, dataList[i]->price()->size(), dataList[i]->fileName());
		size_t size = dataList[i]->price()->size()*sizeof(double);
		gpuErrchk(hipMalloc((void**)&(prices[i]), size));
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(prices[i], dataList[i]->price()->data(), size, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		pSizes[i] = dataList[i]->price()->size();
		totalSize += dataList[i]->price()->size()*sizeof(double)*9 + sizeof(double);
	}
	
	hipDeviceSynchronize();
	gpuErrchk(hipMemcpy(d_pSizes, pSizes, dataList.size()*sizeof(int), hipMemcpyHostToDevice));
	// hipDeviceSynchronize();
	gpuErrchk(hipMemcpy(d_prices, prices, dataList.size()*sizeof(double), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	int * d_status, *status = new int[dataList.size()];
	hipMalloc((void**)&d_status, dataList.size()*sizeof(int));

	hipDeviceSynchronize();
	size_t cudaMem, cudaMemTotal;//, wrapperSize = dataList.size()*sizeof(int);
	// totalSize *= 2;
	// gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemGetInfo(&cudaMem, &cudaMemTotal));
	cout<<"Allocating "<<totalSize<<" bytes on device\n";

	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, totalSize));

	// double* stockPrices, int numDays, double* strikes, double* maturity, bool* call, double* optionPrices, int numOptions

	double **d_optionPrices = 0, **d_strikes = 0, ** optionPrices= new double*[optionList.size()], **strikes = new double*[optionList.size()];
	int *numOptions = 0, *d_numOptions = 0;
	double **d_exp = 0, **exp = new double*[optionList.size()];
	bool** d_call = 0, **call = new bool*[optionList.size()];

	numOptions = new int[optionList.size()];
	hipMalloc((void**)&d_optionPrices, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_exp, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_call, optionList.size()*sizeof(bool*));
	hipMalloc((void**)&d_strikes, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_numOptions, optionList.size()*sizeof(int));
	// cout<<"1\n";
	for(int i = 0; i < optionList.size(); i++){
		optionList[i]->tokenize();
		hipMalloc((void**)&(optionPrices[i]), optionList[i]->call.size()*sizeof(double));
		hipMalloc((void**)&(exp[i]), optionList[i]->exp.size()*sizeof(double));
		hipMalloc((void**)&(call[i]), optionList[i]->call.size()*sizeof(bool));
		hipMalloc((void**)&(strikes[i]), optionList[i]->strike.size()*sizeof(double));

		hipMemcpy(exp[i], optionList[i]->exp.data(), optionList[i]->exp.size()*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(call[i], optionList[i]->call.data(), optionList[i]->call.size()*sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(strikes[i], optionList[i]->strike.data(), optionList[i]->strike.size()*sizeof(double), hipMemcpyHostToDevice);
		numOptions[i] = optionList[i]->call.size();
	}

	// cout<<"2\n";
	// hipMalloc
	gpuErrchk( hipMemcpy(d_exp, exp, optionList.size()*sizeof(double*), hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(d_optionPrices, optionPrices, optionList.size()*sizeof(double*), hipMemcpyHostToDevice));
	hipMemcpy(d_call, call, optionList.size()*sizeof(bool*), hipMemcpyHostToDevice);
	hipMemcpy(d_strikes, strikes, optionList.size()*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_numOptions, numOptions, optionList.size()*sizeof(int), hipMemcpyHostToDevice);

	// printf("OP: %p OP[0]: %p\n", optionPrices, optionPrices[0]);
	// printf("d_OP: %p d_OP[0]: %p\n", d_optionPrices, d_optionPrices[0]);
	// gpuErrchk(hipMalloc((void**)))

	// double** prices, int* sizes, int n, int* status, double** optionPrices, double** strikes, int** exp, bool** calls, int* numOptions

	hipDeviceSynchronize();
	printf("launching kernel\n");
	launch<<<dataList.size()/512+1, 512>>>(d_prices, d_pSizes, dataList.size(), d_status, d_optionPrices, d_strikes, d_exp, d_call, d_numOptions);
	hipDeviceSynchronize();

	// double **temp = new double*[optionList.size()];

	for(int i = 0; i < optionList.size(); i++){
		double *temp = new double[optionList[i]->call.size()];
		// printf("%p\n", d_optionPrices[i]);
		hipMemcpy(temp, optionPrices[i], optionList[i]->call.size()*sizeof(double), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(optionPrices[i]);

		optionPrices[i] = temp;
		optionList[i]->comparePrices(optionPrices[i]);

		hipFree(exp[i]);
		hipFree(strikes[i]);
		hipFree(call[i]);
	}

	hipFree(d_exp);
	hipFree(d_strikes);
	hipFree(d_call);
	hipFree(d_numOptions);
	hipFree(d_optionPrices);
	hipDeviceSynchronize();

	hipMemcpy(status, d_status, dataList.size()*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cout<<"-------------------Long-------------------\n";
	for(int i = 0; i < dataList.size(); i++){
		// cout<<status[i]<<endl;
		if(status[i] == 1)
			cout<<dataList[i]->fileName()<<endl;
	}
	cout<<"-------------------Short-------------------\n";
	for(int i = 0; i < dataList.size(); i++){
		if(status[i] == 2)
			cout<<dataList[i]->fileName()<<endl;
		// cout<<i<<endl;
		hipFree(prices[i]);
	}

	hipFree(d_pSizes);
	hipFree(d_prices);

	// hipFree()
	delete[] optionPrices;
	delete[] exp;
	delete[] strikes;
	delete[] call;
	delete[] prices;
	delete[] pSizes;
	delete[] status;

	return 0;
}

__device__ bool zero(double in){
	double thresh = .002;
	if(in < thresh && in > -thresh){
		// cout<<in<<"~=0\n";
		return true;
	}
	return false;
}

CSVReader::CSVReader(char* fn){
	setFile(fn);
}

void CSVReader::setFile(char* fn){
	file.open(fn);
	filename = fn;//memory leak, causes segfault if initialized from argv. Need to implement copy iterator for strings.
}

vector<char*>* CSVReader::getline(){
	vector<char*>* tokenized;
	if(!this->eof()){
		tokenized = new vector<char*>;//mem leak?
		if(!file.eof()){
			char* line = new char[256];
			// cout<<*line<<endl;

			file.getline(line, 256);
			// cout<<line[0]<<line[1]<<line[2]<<line[3]<<line[4]<<line[5]<<endl;
			tokenized->push_back(&line[0]);
			for(int i = 0; line[i] != '\0'; i++){
				if(line[i]==','){
					line[i] = '\0';
					tokenized->push_back(&line[i+1]);
				}
			}	
		}
	}
	return tokenized;	
}

bool CSVReader::eof(){
	// cout<<"checking eof\n";
	return file.eof();
}

bool CSVReader::isOpen(){
	return file.is_open();
}

CSVReader::~CSVReader(){//eventually add delete[] filename;
	file.close();
}

CSVReader::CSVReader(const CSVReader& copy){
	filename = copy.filename;
	file.open(filename);
}

const char* CSVReader::fileName(){
	return filename;
}

void AVData::tokenize(){
	if(isOpen()){
		while(vector<char*>* tokenized = this->getline()){
			if(!this->eof()){
				// cout<<filename<<endl;
				// cout<<(*tokenized)[1]<<endl;
				if(strcmp((*tokenized)[1], "") != 0)
					open.push_back(stod((*tokenized)[1]));
				if(strcmp((*tokenized)[2], "") != 0)
					high.push_back(stod((*tokenized)[2]));
				if(strcmp((*tokenized)[3], "") != 0)
					low.push_back(stod((*tokenized)[3]));
				if(strcmp((*tokenized)[4], "") != 0)
					close.push_back(stod((*tokenized)[4]));
				if(strcmp((*tokenized)[5], "") != 0)
					volume.push_back(stod((*tokenized)[5]));
			}
			else break;
		}
	}
}

AVData::AVData(char* fn) : CSVReader(fn){}

void addStrToVec(char* str, vector<char*>* v){
	char* temp = new char[strlen(str)+1];
	memcpy(temp, str, strlen(str)+1);
	v->push_back(temp);
}

void listDir(char* dirPath, vector<char*>* v){
	DIR* dirp = opendir(dirPath);
	struct dirent * dp;
	for(int i = 0; (dp = readdir(dirp)) != NULL; i++){
		if(i >= 2) addStrToVec(dp->d_name, v);
		// cout<<temp<<endl;
	}
	closedir(dirp);
}

const vector<double>* AVData::price(){
	return &close;
}

OptionData::OptionData(char* fn) : CSVReader(fn){}

void OptionData::tokenize(){
	if(isOpen()){
		time_t curTime = time(0);
		// cout<<curTime<<endl;
		while(vector<char*>* tokenized = this->getline()){
			if(!this->eof()){
				// cout<<"Reading Line"<<endl;

				call.push_back((strcmp((*tokenized)[0], "C") == 0));
				exp.push_back((stod((*tokenized)[1])+16*3600 - curTime)/(365*24*60*60));
				strike.push_back(stod((*tokenized)[2]));
				bid.push_back(stod((*tokenized)[3]));
				ask.push_back(stod((*tokenized)[4]));
				impVol.push_back(stod((*tokenized)[5]));

			}
			else break;
		}
	}
}

bool* OptionData::comparePrices(double* optionPrices){
	bool* temp = new bool[ask.size()];
	for(int i = 0; i < ask.size(); i++){
		temp[i] = optionPrices[i] < ask[i];
		if(call[i] && temp[i])
			printf("%s %f %f %f %f %f\n", this->fileName(), exp[i], strike[i], optionPrices[i], ask[i], bid[i]);
	}
	return temp;
}